#include "hip/hip_runtime.h"
/*
	CS 6023 Assignment 3. 
	Author: Rupak Paul (CS23M056)
*/

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <vector>



__global__
void applyTranslation(int numTranslations, int *d_TranslationMesh, int *d_TranslationCommand, int *d_TranslationAmount, int *d_TotalMovementInXCoord, int *d_TotalMovementInYCoord) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id < numTranslations) {
		int mesh = d_TranslationMesh[id];
		int command = d_TranslationCommand[id];
		int amount = d_TranslationAmount[id];

		int *iteTotalMovementArr[2] = {&d_TotalMovementInYCoord[mesh], &d_TotalMovementInXCoord[mesh]};
	  	atomicAdd(iteTotalMovementArr[command < 2], (command%2 ? amount : -amount));
	}
}

__global__
void applyTransitiveTranslation(int *d_Offset, int *d_Csr, int *d_WorkListCurr, int *d_WorkListNew, int *d_totalVisitedNode, int *d_TotalMovementInXCoord, int *d_TotalMovementInYCoord, int V) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < V && d_WorkListCurr[id] != -1) {
		int node = id;
		int indexAdjNodes = d_Offset[node];
		int noOfAdjNodes = d_Offset[node+1] - indexAdjNodes;
		atomicAdd(d_totalVisitedNode, 1);

		for(int i = 0; i < noOfAdjNodes; i++) {
			int adj = d_Csr[indexAdjNodes+i];
			d_WorkListNew[adj] = 0;
			d_TotalMovementInXCoord[adj] += d_TotalMovementInXCoord[node];
			d_TotalMovementInYCoord[adj] += d_TotalMovementInYCoord[node]; 
		}
	}
}

__global__
void finalPositionOfMeshes(int *d_TotalMovementInXCoord, int *d_TotalMovementInYCoord, int *d_GlobalCoordinatesX, int *d_GlobalCoordinatesY, int V) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id < V) {
		d_TotalMovementInXCoord[id] += d_GlobalCoordinatesX[id];
		d_TotalMovementInYCoord[id] += d_GlobalCoordinatesY[id];
	}
}

__global__
void computeSceneOpacity(int *d_sceneOpacity, int *d_TotalMovementInXCoord, int *d_TotalMovementInYCoord, int *d_FrameSizeX, int *d_FrameSizeY, int *d_Opacity, int sceneSizeX, int sceneSizeY) {
	int meshId = blockIdx.x;
	int meshXCoord = blockIdx.y;
	int meshYCoord = threadIdx.x;

	if(meshXCoord < d_FrameSizeX[meshId] && meshYCoord < d_FrameSizeY[meshId]) {
		int xPosInScene = meshXCoord + d_TotalMovementInXCoord[meshId];
		int yPosInScene = meshYCoord + d_TotalMovementInYCoord[meshId];

		if(xPosInScene >= 0 && xPosInScene < sceneSizeX && yPosInScene >= 0 && yPosInScene < sceneSizeY) {
			atomicMax(&d_sceneOpacity[xPosInScene*sceneSizeY + yPosInScene], d_Opacity[meshId]);
		}
	}
}

__global__
void computeFinalPNG(int *d_finalPNG, int *d_sceneOpacity, int *d_TotalMovementInXCoord, int *d_TotalMovementInYCoord, int *d_FrameSizeX, int *d_FrameSizeY, int **d_Mesh, int *d_Opacity, int sceneSizeX, int sceneSizeY) {
	int meshId = blockIdx.x;
	int meshXCoord = blockIdx.y;
	int meshYCoord = threadIdx.x;

	if(meshXCoord < d_FrameSizeX[meshId] && meshYCoord < d_FrameSizeY[meshId]) {
		int xPosInScene = meshXCoord + d_TotalMovementInXCoord[meshId];
		int yPosInScene = meshYCoord + d_TotalMovementInYCoord[meshId];

		if(xPosInScene >= 0 && xPosInScene < sceneSizeX && yPosInScene >= 0 && yPosInScene < sceneSizeY) {
			if(d_Opacity[meshId] == d_sceneOpacity[xPosInScene*sceneSizeY + yPosInScene]) {
				int *mesh = d_Mesh[meshId];
				d_finalPNG[xPosInScene*sceneSizeY + yPosInScene] = mesh[meshXCoord*d_FrameSizeY[meshId] + meshYCoord];
			}
		}
	}
}



void copyTranslationFromHostToDevice(int *device_desination, std::vector<std::vector<int>> &translations, int meshCommandAmountFlag) {
	int numOfTranslation = translations.size();
    int *arr = new int[numOfTranslation];
	for(int i = 0; i < numOfTranslation; i++) arr[i] = translations[i][meshCommandAmountFlag];

	hipMemcpy(device_desination, arr, numOfTranslation*sizeof(int), hipMemcpyHostToDevice);
    delete arr;
}

void copyMesheshFromHostToDevice(int **hMesh, int **d_Mesh, int *hFrameSizeX, int *hFrameSizeY, int V) {
	int **meshPtrs = new int*[V];
    
	for(int i = 0; i < V; i++) {
		hipMalloc(&meshPtrs[i], hFrameSizeX[i]*hFrameSizeY[i]*sizeof(int));
		hipMemcpy(meshPtrs[i], hMesh[i], hFrameSizeX[i]*hFrameSizeY[i]*sizeof(int), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_Mesh, meshPtrs, V*sizeof(int*), hipMemcpyHostToDevice);
    delete meshPtrs;
}



void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input. 
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;
	

	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ; 
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ; 
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}

void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL; 
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}
	
	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}



int main (int argc, char **argv) {
	
	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ; 

	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;
	
	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;  
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now () ;

	// Code begins here.
		
	int *d_TotalMovementInXCoord;
	int *d_TotalMovementInYCoord;

	hipMalloc(&d_TotalMovementInXCoord, V*sizeof(int));
	hipMalloc(&d_TotalMovementInYCoord, V*sizeof(int));
	hipMemset(d_TotalMovementInXCoord, 0, V*sizeof(int));
	hipMemset(d_TotalMovementInYCoord, 0, V*sizeof(int));



	int *d_TranslationMesh;
	int *d_TranslationCommand;
	int *d_TranslationAmount;
	
	hipMalloc(&d_TranslationMesh, numTranslations*sizeof(int));
	hipMalloc(&d_TranslationCommand, numTranslations*sizeof(int));
	hipMalloc(&d_TranslationAmount, numTranslations*sizeof(int));
	copyTranslationFromHostToDevice(d_TranslationMesh, translations, 0);
	copyTranslationFromHostToDevice(d_TranslationCommand, translations, 1);
	copyTranslationFromHostToDevice(d_TranslationAmount, translations, 2);

	applyTranslation<<<int(ceil(numTranslations / 1024.0)), 1024>>>(numTranslations, d_TranslationMesh, d_TranslationCommand, d_TranslationAmount, d_TotalMovementInXCoord, d_TotalMovementInYCoord);
	hipDeviceSynchronize();

	hipFree(d_TranslationMesh);
	hipFree(d_TranslationCommand);
	hipFree(d_TranslationAmount);



	int *d_WorkListCurr;
	int *d_WorkListNew;
	int *d_totalVisitedNode;
	int *d_Offset;
	int *d_Csr;
	
	hipMalloc(&d_WorkListCurr, V*sizeof(int));
	hipMalloc(&d_WorkListNew, V*sizeof(int));
	hipMalloc(&d_totalVisitedNode, sizeof(int));
	hipMalloc(&d_Offset, (V+1)*sizeof(int));
	hipMalloc(&d_Csr, E*sizeof(int));
	hipMemset(d_WorkListCurr, -1, V*sizeof(int));
	hipMemset(d_WorkListNew, -1, V*sizeof(int));
	hipMemset(d_WorkListCurr, 0, sizeof(int));
	hipMemset(d_totalVisitedNode, 0, sizeof(int));
	hipMemcpy(d_Offset, hOffset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Csr, hCsr, E*sizeof(int), hipMemcpyHostToDevice);
	
	while(true) {
		applyTransitiveTranslation<<<int(ceil(V / 1024.0)), 1024>>>(d_Offset, d_Csr, d_WorkListCurr, d_WorkListNew, d_totalVisitedNode, d_TotalMovementInXCoord, d_TotalMovementInYCoord, V);
		hipDeviceSynchronize();
		
		std::swap(d_WorkListCurr, d_WorkListNew);
		
		int totalVisitedNode;
		hipMemcpy(&totalVisitedNode, d_totalVisitedNode, sizeof(int), hipMemcpyDeviceToHost);
		if(totalVisitedNode == V) break;
		else hipMemset(d_WorkListNew, -1, V*sizeof(int));
	}

	hipFree(d_WorkListCurr);
	hipFree(d_WorkListNew);
	hipFree(d_totalVisitedNode);
	hipFree(d_Offset);
	hipFree(d_Csr);



	int *d_GlobalCoordinatesX;
	int *d_GlobalCoordinatesY;

	hipMalloc(&d_GlobalCoordinatesX, V*sizeof(int));
	hipMalloc(&d_GlobalCoordinatesY, V*sizeof(int));
	hipMemcpy(d_GlobalCoordinatesX, hGlobalCoordinatesX, V*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_GlobalCoordinatesY, hGlobalCoordinatesY, V*sizeof(int), hipMemcpyHostToDevice);

	finalPositionOfMeshes<<<int(ceil(V / 1024.0)), 1024>>>(d_TotalMovementInXCoord, d_TotalMovementInYCoord, d_GlobalCoordinatesX, d_GlobalCoordinatesY, V);
	hipDeviceSynchronize();

	hipFree(d_GlobalCoordinatesX);
	hipFree(d_GlobalCoordinatesY);



	int *d_finalPNG;
	int *d_sceneOpacity;
	int *d_FrameSizeX;
	int *d_FrameSizeY;
	int *d_Opacity;
	int **d_Mesh;

	hipMalloc(&d_finalPNG, frameSizeX*frameSizeY*sizeof(int));
	hipMalloc(&d_sceneOpacity, frameSizeX*frameSizeY*sizeof(int));
	hipMalloc(&d_FrameSizeX, V*sizeof(int));
	hipMalloc(&d_FrameSizeY, V*sizeof(int));
	hipMalloc(&d_Opacity, V*sizeof(int));
	hipMalloc(&d_Mesh, V*sizeof(int*));
	hipMemset(d_finalPNG, 0, frameSizeX*frameSizeY*sizeof(int));
	hipMemset(d_sceneOpacity, -1, frameSizeX*frameSizeY*sizeof(int));
	hipMemcpy(d_FrameSizeX, hFrameSizeX, V*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_FrameSizeY, hFrameSizeY, V*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Opacity, hOpacity, V*sizeof(int), hipMemcpyHostToDevice);
	copyMesheshFromHostToDevice(hMesh, d_Mesh, hFrameSizeX, hFrameSizeY, V);

	computeSceneOpacity<<<dim3(V, 100, 1), 100>>>(d_sceneOpacity, d_TotalMovementInXCoord, d_TotalMovementInYCoord, d_FrameSizeX, d_FrameSizeY, d_Opacity, frameSizeX, frameSizeY);

	computeFinalPNG<<<dim3(V, 100, 1), 100>>>(d_finalPNG, d_sceneOpacity, d_TotalMovementInXCoord, d_TotalMovementInYCoord, d_FrameSizeX, d_FrameSizeY, d_Mesh, d_Opacity, frameSizeX, frameSizeY);
	
	hipDeviceSynchronize();
	hipMemcpy(hFinalPng, d_finalPNG, frameSizeX*frameSizeY*sizeof(int), hipMemcpyDeviceToHost);

	// Code ends here.

	auto end  = std::chrono::high_resolution_clock::now () ;

	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken) ;
	// Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;	

}
